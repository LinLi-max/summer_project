#include "hip/hip_runtime.h"
#include "kernel.h"

//initialization
__global__ void init_kernel(double* d_U, double dx, double dy, double gam, int col, int row)
{
    unsigned long long tid = (blockIdx.x * blockDim.x + threadIdx.x);

    const double rou1 = 1.0, u1 = 2.9, v1 = 0.0, p1 = 0.71429;  //uniform inlet condition at left boundary
    const double rou2 = 1.69997, u2 = 2.61934, v2 = -0.50632, p2 = 1.52819;  //uniform inlet condition at up boundary
    const double pi = 3.141592654, alpha = 29 * pi / 180;  //incidence radian of oblique shock wave

    if (tid >= col * row)
    {
        return;
    }
        
    int size = col * row;
    int  xx = tid % col;
    int  yy = tid / col;

    double x = (1 - yy * dy) / tan(alpha);

    if (xx * dx <= x)
    {
        d_U[tid] = rou1;
        d_U[tid + size] = rou1 * u1;
        d_U[tid + size * 2] = rou1 * v1;
        d_U[tid + size * 3] = p1 / (gam - 1) + rou1 * (u1 * u1 + v1 * v1) / 2;
    }
    else
    {
        d_U[tid] = rou2;
        d_U[tid + size] = rou2 * u2;
        d_U[tid + size * 2] = rou2 * v2;
        d_U[tid + size * 3] = p2 / (gam - 1) + rou2 * (u2 * u2 + v2 * v2) / 2;
    }

}

void call_init(double* d_U, double dx, double dy, double gam, int col, int row, int TPB)
{
    int BSIZE = (col * row + (TPB - 1)) / TPB;
    init_kernel << <BSIZE, TPB >> > (d_U, dx, dy, gam, col, row);
    hipDeviceSynchronize();
}

//calculation of dt based on the algorithm stability parameter cfl
__global__ void cfl_kernel(double* d_U, double dx, double dy, double* dt, double gam, double sf, int col, int row)
{
    unsigned long long tid = (blockIdx.x * blockDim.x + threadIdx.x);
    double maxvel = 0;

    int xx = tid % col;
    int yy = tid / col;
    int size = col * row;

    if (xx < 1 || xx >= col || yy < 1 || yy >= row)
    {
        return;
    }
        
    double u = d_U[tid + size] / d_U[tid];
    double v = d_U[tid + size * 2] / d_U[tid];
    double p = (gam - 1) * (d_U[tid + size * 3] - 0.5 * d_U[tid] * (u * u + v * v));
    double velocity = sqrt(gam * p / d_U[tid]) + sqrt(u * u + v * v);  //feature speed
    if (velocity > maxvel)
    {
        maxvel = velocity;
    }

    dt[0] = sf * min(dx, dy) / maxvel;
}

void call_cfl(double* d_U, double dx, double dy, double* dt, double sf, double gam, int col, int row)
{

    double maxvel = 0;
    long int size = col * row;

    for (int i = 1; i <= gx; i++)
    {
        for (int j = 1; j <= gy; j++)
        {
            double u0 = d_U[i + j * col];
            double u1 = d_U[i + j * col + size];
            double u2 = d_U[i + j * col + size * 2];
            double u3 = d_U[i + j * col + size * 3];

            double u = u1 / u0;
            double v = u2 / u0;
            double p = (gam - 1) * (u3 - 0.5 * u0 * (u * u + v * v));
            double velocity = sqrt(gam * p / u0) + sqrt(u * u + v * v);  //feature speed
            if (velocity > maxvel)
            {
                maxvel = velocity;
            }
        }
    }

    *dt = sf * min(dx, dy) / maxvel;

}

//deal with the boundary
__global__ void bound_kernel(double* d_U, double gam, int col, int row)
{
    const double rou1 = 1.0, u1 = 2.9, v1 = 0.0, p1 = 0.71429;  //uniform inlet condition at left boundary
    const double rou2 = 1.69997, u2 = 2.61934, v2 = -0.50632, p2 = 1.52819;  //uniform inlet condition at up boundary

    unsigned long long tid = (blockIdx.x * blockDim.x + threadIdx.x);
    int xx = tid % col;
    int yy = tid / col;
    int size = col * row;

    if (xx >= col || yy >= row)
    {
        return;
    }
        
    //left
    if (xx == 0 && yy <= gy + 1)
    {
        d_U[tid] = rou1;
        d_U[tid + size] = rou1 * u1;
        d_U[tid + size * 2] = rou1 * v1;
        d_U[tid + size * 3] = p1 / (gam - 1) + rou1 * (u1 * u1 + v1 * v1) / 2;
    }

    //right
    if (xx == gx + 1 && yy <= gy + 1)
    {
        for (int k = 0; k < 4; k++)
        {
            d_U[tid + k * size] = d_U[gx + yy * col + k * size];
        }
    }

    //up 
    if (yy == gy + 1 && xx <= gx + 1)
    {
        d_U[tid] = rou2;
        d_U[tid + size] = rou2 * u2;
        d_U[tid + size * 2] = rou2 * v2;
        d_U[tid + size * 3] = p2 / (gam - 1) + rou2 * (u2 * u2 + v2 * v2) / 2;
    }

    //down
    if (yy == 0 && xx <= gx + 1)
    {
        d_U[tid] = d_U[xx + 1 * col];
        d_U[tid + size] = d_U[xx + 1 * col + size];
        d_U[tid + size * 2] = 0;
        d_U[tid + size * 3] = d_U[xx + 1 * col + size * 3];
    }
}

void call_bound(double* d_U, double gam, int col, int row, int TPB)
{
    int BSIZE = (col * row + (TPB - 1)) / TPB;
    bound_kernel << <BSIZE, TPB >> > (d_U, gam, col, row);
    hipDeviceSynchronize();
}

//differential in x-direction
__global__ void updataU_kernel(double* d_U, double* temp, double dx, double dy, double dt, double gam, int col, int row)
{
    const int a = 3.0;  //speed of sound not exceeding 3
    double eta = (a * dt / dx) * (1 - a * dt / dx);

    unsigned long long tid = (blockIdx.x * blockDim.x + threadIdx.x);

    int xx = tid % col;
    int yy = tid / col;
    int size = col * row;
 
    if (xx < 1 || xx > gx || yy < 0 || yy > gy + 1)
    {
        return;
    }
        
    //switching function
    double theta = fabs(fabs(d_U[xx + 1 + yy * col] - d_U[tid]) - fabs(d_U[tid] - d_U[xx - 1 + yy * col]))
        / (fabs(d_U[xx + 1 + yy * col] - d_U[tid]) + fabs(d_U[tid] - d_U[xx - 1 + yy * col]) + 1e-100);

    for (int k = 0; k < 4; k++)
    {
        temp[tid + k * size] = d_U[tid + k * size] + 0.5 * eta * theta * (d_U[xx + 1 + yy * col + k * size]
            - 2 * d_U[tid + k * size] + d_U[xx - 1 + yy * col + k * size]);
    }
}

__global__ void updataU2_kernel(double* d_U, double* d_F, double dx, double dy, double dt, double gam, int col, int row)
{
    unsigned long long tid = (blockIdx.x * blockDim.x + threadIdx.x);

    int xx = tid % col;
    int yy = tid / col;
    int size = col * row;

    if (xx < 1 || xx > gx || yy < 0 || yy > gy + 1)
    {
        return;
    }
        
    for (int i = 0; i < 4; i++)
    {
        d_U[tid + i * size] = d_U[tid + i * size] - (dt / dx) *
            (d_F[tid + i * size] - d_F[xx - 1 + yy * col + i * size]);

    }
}

__global__ void temp2U_kernel(double* d_U, double* temp, double dx, double dy, double dt, double gam, int col, int row)
{
    unsigned long long tid = (blockIdx.x * blockDim.x + threadIdx.x);

    int xx = tid % col;
    int yy = tid / col;
    int size = col * row;

    if (xx < 1 || xx > gx || yy < 0 || yy > gy + 1)
    {
        return;
    }
        
    for (int k = 0; k < 4; k++)
    {
        d_U[tid + k * size] = temp[tid + k * size];
    }
}

__global__ void updataF_kernel(double* d_U, double* d_F, double dx, double dy, double dt, double gam, int col, int row)
{
    unsigned long long tid = (blockIdx.x * blockDim.x + threadIdx.x);

    int xx = tid % col;
    int yy = tid / col;
    int size = col * row;

    if (xx < 0 || xx > gx + 1 || yy < 0 || yy > gy + 1)
    {
        return;
    }
        
    double* u0 = d_U;
    double* u1 = d_U + size;
    double* u2 = d_U + 2 * size;
    double* u3 = d_U + 3 * size;

    double u = u1[tid] / u0[tid];
    double v = u2[tid] / u0[tid];
    double p = (gam - 1) * (u3[tid] - 0.5 * u0[tid] * (u * u + v * v));

    d_F[tid] = u1[tid];
    d_F[tid + size] = u0[tid] * u * u + p;
    d_F[tid + size * 2] = u0[tid] * u * v;
    d_F[tid + size * 3] = (u3[tid] + p) * u;
}

__global__ void updataF_kernel2(double* d_U, double* d_F, double dx, double dy, double dt, double gam, int col, int row)
{
    unsigned long long tid = (blockIdx.x * blockDim.x + threadIdx.x);

    int xx = tid % col;
    int yy = tid / col;
    int size = col * row;

    if (xx < 0 || xx > gx || yy < 0 || yy > gy + 1)
    {
        return;
    }
        
    double* u0 = d_U;
    double* u1 = d_U + size;
    double* u2 = d_U + 2 * size;
    double* u3 = d_U + 3 * size;

    double u = u1[tid] / u0[tid];
    double v = u2[tid] / u0[tid];
    double p = (gam - 1) * (u3[tid] - 0.5 * u0[tid] * (u * u + v * v));

    d_F[tid] = u1[tid];
    d_F[tid + size] = u0[tid] * u * u + p;
    d_F[tid + size * 2] = u0[tid] * u * v;
    d_F[tid + size * 3] = (u3[tid] + p) * u;
}

__global__ void updataUhalf_kernel(double* d_U, double* d_U_half, double* d_F, double dx, double dy, double dt, double gam, int col, int row)
{
    unsigned long long tid = (blockIdx.x * blockDim.x + threadIdx.x);

    int xx = tid % col;
    int yy = tid / col;
    int size = col * row;

    if (xx < 0 || xx > gx || yy < 0 || yy > gy + 1)
    {
        return;
    }
       
    for (int i = 0; i < 4; i++)
    {
        d_U_half[tid + i * size] = 0.5 * (d_U[xx + 1 + yy * col + i * size]
            + d_U[tid + i * size]) - 0.5 * dt / dx * (d_F[xx + 1 + yy * col + i * size]
                - d_F[tid + i * size]);

    }
}

//differential in y-direction
__global__ void updataU_kernel_y(double* d_U, double* temp, double dx, double dy, double dt, double gam, int col, int row)
{
    const int a = 3.0;  //speed of sound not exceeding 3
    double eta = (a * dt / dx) * (1 - a * dt / dx);

    unsigned long long tid = (blockIdx.x * blockDim.x + threadIdx.x);

    int xx = tid % col;
    int yy = tid / col;
    int size = col * row;

    if (xx < 0 || xx > gx + 1 || yy < 1 || yy > gy)
    {
        return;
    }
       
    //switching function
    double theta = fabs(fabs(d_U[xx + (yy + 1) * col] - d_U[tid]) - fabs(d_U[tid] - d_U[xx + (yy - 1) * col]))
        / (fabs(d_U[xx + (yy + 1) * col] - d_U[tid]) + fabs(d_U[tid] - d_U[xx + (yy - 1) * col]) + 1e-100);

    for (int k = 0; k < 4; k++)
    {
        temp[tid + k * size] = d_U[tid + k * size] + 0.5 * eta * theta * (d_U[xx + (yy + 1) * col + k * size]
            - 2 * d_U[tid + k * size] + d_U[xx + (yy - 1) * col + k * size]);
    }
}

__global__ void updataU2_kernel_y(double* d_U, double* d_G, double dx, double dy, double dt, double gam, int col, int row)
{
    unsigned long long tid = (blockIdx.x * blockDim.x + threadIdx.x);

    int xx = tid % col;
    int yy = tid / col;
    int size = col * row;

    if (xx < 0 || xx > gx + 1 || yy < 1 || yy > gy)
    {
        return;
    }
        
    for (int i = 0; i < 4; i++)
    {
        d_U[tid + i * size] = d_U[tid + i * size] - (dt / dx) *
            (d_G[tid + i * size] - d_G[xx + (yy - 1) * col + i * size]);

    }
}

__global__ void temp2U_kernel_y(double* d_U, double* temp, double dx, double dy, double dt, double gam, int col, int row)
{
    unsigned long long tid = (blockIdx.x * blockDim.x + threadIdx.x);

    int xx = tid % col;
    int yy = tid / col;
    int size = col * row;

    if (xx < 0 || xx > gx + 1 || yy < 1 || yy > gy)
    {
        return;
    }
        
    for (int k = 0; k < 4; k++)
    {
        d_U[tid + k * size] = temp[tid + k * size];
    }
}

__global__ void updataG_kernel_y(double* d_U, double* d_G, double dx, double dy, double dt, double gam, int col, int row)
{
    unsigned long long tid = (blockIdx.x * blockDim.x + threadIdx.x);

    int xx = tid % col;
    int yy = tid / col;
    int size = col * row;

    if (xx < 0 || xx > gx + 1 || yy < 0 || yy > gy + 1)
    {
        return;
    }
        
    double* u0 = d_U;
    double* u1 = d_U + size;
    double* u2 = d_U + 2 * size;
    double* u3 = d_U + 3 * size;

    double u = u1[tid] / u0[tid];
    double v = u2[tid] / u0[tid];
    double p = (gam - 1) * (u3[tid] - 0.5 * u0[tid] * (u * u + v * v));

    d_G[tid] = u2[tid];
    d_G[tid + size] = u0[tid] * u * v;
    d_G[tid + size * 2] = u0[tid] * v * v + p;
    d_G[tid + size * 3] = (u3[tid] + p) * v;
}

__global__ void updataG_kernel2_y(double* d_U, double* d_G, double dx, double dy, double dt, double gam, int col, int row)
{
    unsigned long long tid = (blockIdx.x * blockDim.x + threadIdx.x);

    int xx = tid % col;
    int yy = tid / col;
    int size = col * row;

    if (xx < 0 || xx > gx + 1 || yy < 0 || yy > gy)
    {
        return;
    }
        
    double* u0 = d_U;
    double* u1 = d_U + size;
    double* u2 = d_U + 2 * size;
    double* u3 = d_U + 3 * size;

    double u = u1[tid] / u0[tid];
    double v = u2[tid] / u0[tid];
    double p = (gam - 1) * (u3[tid] - 0.5 * u0[tid] * (u * u + v * v));

    d_G[tid] = u2[tid];
    d_G[tid + size] = u0[tid] * u * v;
    d_G[tid + size * 2] = u0[tid] * v * v + p;
    d_G[tid + size * 3] = (u3[tid] + p) * v;
}

__global__ void updataUhalf_kernel_y(double* d_U, double* d_U_half, double* d_G, double dx, double dy, double dt, double gam, int col, int row)
{
    unsigned long long tid = (blockIdx.x * blockDim.x + threadIdx.x);

    int xx = tid % col;
    int yy = tid / col;
    int size = col * row;

    if (xx < 0 || xx > gx + 1 || yy < 0 || yy > gy)
    {
        return;
    }
        
    for (int i = 0; i < 4; i++)
    {
        d_U_half[tid + i * size] = 0.5 * (d_U[xx + (yy + 1) * col + i * size]
            + d_U[tid + i * size]) - 0.5 * dt / dx * (d_G[xx + (yy + 1) * col + i * size]
                - d_G[tid + i * size]);
    }
}

//Lax-Wendroff 2d sovler
void call_solve_x(double* d_U, double* d_U_half, double* d_FG, double* d_temp, double dx, double dy, double dt, double gam, int col, int row, int TPB)
{
    int BSIZE = (col * row + (TPB - 1)) / TPB;

    hipMemset(d_temp, 0, sizeof(double) * col * row * 4);

    updataU_kernel << <BSIZE, TPB >> > (d_U, d_temp, dx, dy, dt, gam, col, row);

    temp2U_kernel << <BSIZE, TPB >> > (d_U, d_temp, dx, dy, dt, gam, col, row);

    updataF_kernel << <BSIZE, TPB >> > (d_U, d_FG, dx, dy, dt, gam, col, row);

    updataUhalf_kernel << <BSIZE, TPB >> > (d_U, d_U_half, d_FG, dx, dy, dt, gam, col, row);

    updataF_kernel2 << <BSIZE, TPB >> > (d_U_half, d_FG, dx, dy, dt, gam, col, row);

    updataU2_kernel << <BSIZE, TPB >> > (d_U, d_FG, dx, dy, dt, gam, col, row);
}

void call_solve_y(double* d_U, double* d_U_half, double* d_FG, double* d_temp, double dx, double dy, double dt, double gam, int col, int row, int TPB)
{
    int BSIZE = (col * row + (TPB - 1)) / TPB;

    hipMemset(d_temp, 0, sizeof(double) * col * row * 4);

    updataU_kernel_y << <BSIZE, TPB >> > (d_U, d_temp, dx, dy, dt, gam, col, row);

    temp2U_kernel_y << <BSIZE, TPB >> > (d_U, d_temp, dx, dy, dt, gam, col, row);

    updataG_kernel_y << <BSIZE, TPB >> > (d_U, d_FG, dx, dy, dt, gam, col, row);

    updataUhalf_kernel_y << <BSIZE, TPB >> > (d_U, d_U_half, d_FG, dx, dy, dt, gam, col, row);

    updataG_kernel2_y << <BSIZE, TPB >> > (d_U_half, d_FG, dx, dy, dt, gam, col, row);

    updataU2_kernel_y << <BSIZE, TPB >> > (d_U, d_FG, dx, dy, dt, gam, col, row);
}

void call_solver_2d(double* d_U, double* d_U_half, double* d_FG, double* d_temp, double dx, double dy, double dt, double gam, int col, int row, int TPB)
{
    call_solve_x(d_U, d_U_half, d_FG, d_temp, dx, dy, dt / 2.0, gam, col, row, TPB);
    call_bound(d_U, gam, col, row, TPB);

    call_solve_y(d_U, d_U_half, d_FG, d_temp, dx, dy, dt / 2.0, gam, col, row, TPB);
    call_bound(d_U, gam, col, row, TPB);

    call_solve_y(d_U, d_U_half, d_FG, d_temp, dx, dy, dt / 2.0, gam, col, row, TPB);
    call_bound(d_U, gam, col, row, TPB);

    call_solve_x(d_U, d_U_half, d_FG, d_temp, dx, dy, dt / 2.0, gam, col, row, TPB);
    call_bound(d_U, gam, col, row, TPB);
}
