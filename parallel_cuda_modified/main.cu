#include "hip/hip_runtime.h"
/**
 * \file main.c
 * \brief The main function of oblique shock reflection simulation(cuda version).
 * \author Lin li
 * \version 2.0
 * \date 2021-07-29
 */

#include <stdio.h>
#include <time.h>
#include "kernel.h"

int output(double* d_U, double dx, double dy, double gam, int col, int row);

int main(int argc, char* argv[])
{
    //calculation area
    const double x = 4.0, y = 1.0;

    //physical constants
    const double sf = 0.8, gam = 1.4; 
    
    //time of simulation
    const double total_time = 3.0;

    clock_t start, finish;
    double runtime;
    double current_time = 0;
    const double dx = x / gx;
    const double dy = y / gy;
    int TPB = 128;
    int col = gx + 2;
    int row = gy + 2;
    double* dt, * d_temp;
    double* d_U, * d_U_half, * d_FG;
    double* d_U_cpu, *dt_cpu;

    if (argc != 2)
    {
        printf("Please input TPB\n");
        return -1;
    }

    TPB = atoi(argv[1]);
    printf("TPB = %d\n", TPB);
    printf("gx = %d, gy = %d\n", gx, gy);
    printf("Simulation time = %.3lf seconds.\n", total_time);

    //hipMallocManaged((void**)&d_U, sizeof(double) * col * row * 4);
    //hipMallocManaged((void**)&d_U_half, sizeof(double) * col * row * 4);
    //hipMallocManaged((void**)&d_FG, sizeof(double) * col * row * 4);
    //hipMallocManaged((void**)&d_temp, sizeof(double) * col * row * 4);
    //hipMallocManaged((void**)&dt, sizeof(double));

    hipMalloc((void**)&d_U, sizeof(double) * col * row * 4);
	d_U_cpu = (double*)malloc(sizeof(double) * col * row * 4);

    hipMalloc((void**)&d_U_half, sizeof(double) * col * row * 4);
    hipMalloc((void**)&d_FG, sizeof(double) * col * row * 4);
    hipMalloc((void**)&d_temp, sizeof(double) * col * row * 4);

    hipMalloc((void**)&dt, sizeof(double));
	dt_cpu = (double*)malloc(sizeof(double));

    start = clock();

    call_init(d_U, dx, dy, gam, col, row, TPB);  //initialization on the GPU

	// Need to bring d_U and dt back to the host before calling call_cfl
	hipMemcpy(d_U_cpu, d_U, sizeof(double) * col * row * 4, hipMemcpyDeviceToHost);
	hipMemcpy(dt_cpu, dt, sizeof(double), hipMemcpyDeviceToHost);

    call_cfl(d_U_cpu, dx, dy, dt_cpu, sf, gam, col, row);  //calculate dt using the cfl number - on the CPU

	// Need to bring dt to the GPU before calling the main loop
	hipMemcpy(dt, dt_cpu, sizeof(double), hipMemcpyHostToDevice);

    while (current_time < total_time)
    {
        call_solver_2d(d_U, d_U_half, d_FG, d_temp, dx, dy, dt_cpu[0], gam, col, row, TPB);  //use a second order Lax-Wendroff two-step difference scheme
        current_time = current_time + *dt_cpu;
        //printf("time = %10g\n", current_time);  //the simulation process should be commented out when testing the runing time  
    }

    finish = clock();

    runtime = (finish - start) / CLOCKS_PER_SEC;
    printf("Program done, the cuda version use %.3lf seconds.\n", runtime);

	// Need to bring d_U back to the host before calling output
	hipMemcpy(d_U_cpu, d_U, sizeof(double) * col * row * 4, hipMemcpyDeviceToHost);
    output(d_U_cpu, dx, dy, gam, col, row);

    hipFree(d_U);
    hipFree(d_U_half);
    hipFree(d_FG);
    hipFree(d_temp);
    hipFree(dt);

	free(d_U_cpu);
	free(dt_cpu);

    return 0;
}

int output(double* d_U, double dx, double dy, double gam, int col, int row)
{
    FILE* fp;
    double rou, u, v, p;

    //for density contour map
    fp = fopen("result.txt", "w");
    if (fp == NULL)
    {
        perror("Error opening result.txt for writing");
        return(-1);
    }

    fprintf(fp, "variables = x, y, rou, u, v, p, E\n");
    fprintf(fp, "gx = %d, gy = %d\n", gx, gy);

    long int size = col * row;
    for (int i = 1; i <= gx; i++)
    {
        for (int j = 1; j <= gy; j++)
        {
            rou = d_U[i + col * j];
            u = d_U[i + col * j + size] / rou;
            v = d_U[i + col * j + size * 2] / rou;
            p = (gam - 1) * (d_U[i + col * j + size * 3] - 0.5 * d_U[i + col * j] * (u * u + v * v));
            fprintf(fp, "%10lf%10lf%10lf%10lf%10lf%10lf%10lf\n", i * dx, j * dy, rou, u, v, p, d_U[i + col * j + size * 3]);
        }
    }
    fclose(fp);

    //for pressure map at position y / 2
    fp = fopen("result_0.5y.txt", "w");
    if (fp == NULL)
    {
        perror("Error opening result_0.5y.txt for writing");
        return(-1);
    }

    fprintf(fp, "variables = x, rou, u, v, p, E\n");
    fprintf(fp, "gx = %d, gy = %d, 0.5y position\n", gx, gy);

    int j = gy / 2;
    for (int i = 1; i <= gx; i++)
    {
        rou = d_U[i + col * j];
        u = d_U[i + col * j + size] / rou;
        v = d_U[i + col * j + size * 2] / rou;
        p = (gam - 1) * (d_U[i + col * j + size * 3] - 0.5 * d_U[i + col * j] * (u * u + v * v));
        fprintf(fp, "%10lf%10lf%10lf%10lf%10lf%10lf\n", i * dx, rou, u, v, p, d_U[i + col * j + size * 3]);
    }
    fclose(fp);

    return 0;
}
